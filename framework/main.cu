#include "hip/hip_runtime.h"
#include "util.h"
#include "usage.h"
#include "image.h"
#include "gradient.h"
#include "kernels.h"
#include <iostream>
using namespace std;

// Texture memory for the G values
texture<float, 2, hipReadModeElementType> texRef;

/**
 * Calculate the depth from the disparity values
 *
 * Z = baseline * f / (d + doffs)
 * baseline: 	camera baseline in mm
 * f: 			focal length in pixels
 * d:			disparity for pixel
 * doffs:		x-difference of principal points (cx1 - cx0) for im1 and im0
 */
__global__ void g_compute_depth(float * Disparities, float *Depths, int w,
		int h, float baseline, int f, int doffs)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x < w && y < h)
	{
		float d = read_data(Disparities, w, h, x, y);
		write_data(Depths, baseline * f / (d + doffs), w, h, x, y);
	}
}

/**
 * Compute the alpha parameter for the circle of confusion
 *
 * alpha = F²/(n * (Zf - F)) with focal length F and aperture n
 * and depth of the focal plane between gamma_min, gamma_max = [0, 1]
 */
__device__ __host__ float alpha(float f, float n, int z_f)
{
	return square(f) / (n * (z_f - f));
}

/**
 * Compute the g function via an estimated circle of confusion
 *
 * See: Bertalmio, Fort et al: Real-time, Accurate Depth of Field
 * using Anisotropic Diffusion and Programmable Graphic Cards
 * http://www.dtic.upf.edu/~mbertalmio/dof/dof01.pdf
 *
 * g(x,y) = alpha * (| Z(x,y) - Zf | / Z(x,y) )²
 *
 * z_f is the depth of the focal plane between gamma_min, gamma_max = [0,1]
 * g needs to be scaled in order to lie between [0,1]
 */
__global__ void g_compute_g(float *Depths, float *G, int w, int h, float z_f,
		float alpha)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	float z;

	if (x < w && y < h)
	{
		z = read_data(Depths, w, h, x, y);
		write_data(G, alpha * square(fabs(z - z_f) / z), w, h, x, y);
	}
}

/**
 * Apply G and write the result back to the gradient vectors
 */
__global__ void g_apply_g(float *Grad_x, float *Grad_y, int w, int h, int nc)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x < w && y < h)
	{
		float g = tex2D(texRef, x + 0.5f, y + 0.5f);
		float grad_x, grad_y;

		for (int c = 0; c < nc; c++)
		{
			grad_x = read_data(Grad_x, w, h, nc, x, y, c);
			grad_y = read_data(Grad_y, w, h, nc, x, y, c);

			// Write v_1 back to V_1 and v_2 to V_2
			write_data(Grad_x, g * grad_x, w, h, nc, x, y, c);
			write_data(Grad_y, g * grad_y, w, h, nc, x, y, c);
		}
	}

}

/**
 * Compute the update step as In+1 = In + tau*D
 */
__global__ void g_update_step(float *D, float *I, int w, int h, int nc,
		float tau)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int c = threadIdx.z + blockDim.z * blockIdx.z;

	float upd;

	if (x < w && y < h && c < nc)
	{
		upd = read_data(I, w, h, nc, x, y, c)
				+ tau * read_data(D, w, h, nc, x, y, c);
		write_data(I, upd, w, h, nc, x, y, c);
	}
}


cv::Mat calculate_disparities(const config c)
{
	// define the range of gamma
	int gc = c.gamma_max - c.gamma_min + 1;

	// image + 0 is left
	string imageL = c.image + "0.png";
	// image + 1 is right
	string imageR = c.image + "1.png";

	cv::Mat mInL = load_image(imageL, c.gray);
	cv::Mat mInR = load_image(imageR, c.gray);

	// Width, height and channels of image
	int w, h, nc;
	get_dimensions(mInL, mInR, w, h, nc);
	cout << "Image Dimensions: " << w << " x " << h << " x " << nc << endl;

	// Define output array, grayscale image of depth values
	cv::Mat mOut(h, w, CV_32FC1);

	// allocate raw input image array
	float *imgInL = new float[(size_t) w * h * nc];
	float *imgInR = new float[(size_t) w * h * nc];

	// allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
	float *imgOut = new float[(size_t) w * h * mOut.channels()];

	// Init raw input image array
	convert_mat_to_layered(imgInL, mInL);
	convert_mat_to_layered(imgInR, mInR);

	// Allocate memory on device for images
	size_t imgBytes = w * h * nc * sizeof(float);
	float *IL, *IR = NULL;
	hipMalloc(&IL, imgBytes);
	CUDA_CHECK;
	hipMalloc(&IR, imgBytes);
	CUDA_CHECK;
	hipMemset(IL, 0, imgBytes);
	CUDA_CHECK;
	hipMemset(IR, 0, imgBytes);
	CUDA_CHECK;

	// for P (3 channels for p1..3) and Phi vectors
	size_t pBytes = w * h * gc * 3 * sizeof(float);
	size_t phiBytes = w * h * gc * sizeof(float);
	float *P, *Phi = NULL;
	hipMalloc(&P, pBytes);
	CUDA_CHECK;
	hipMalloc(&Phi, phiBytes);
	CUDA_CHECK;
	hipMemset(P, 0, pBytes);
	CUDA_CHECK;
	hipMemset(Phi, 0, phiBytes);
	CUDA_CHECK;

	// for grad3 of phi and div3 of p
	float * Grad3_Phi, *Div3_P = NULL;
	hipMalloc(&Grad3_Phi, pBytes);
	CUDA_CHECK;
	hipMalloc(&Div3_P, phiBytes);
	CUDA_CHECK;
	hipMemset(Grad3_Phi, 0, pBytes);
	CUDA_CHECK;
	hipMemset(Div3_P, 0, phiBytes);
	CUDA_CHECK;

	// for the final depth values
	size_t gBytes = w * h * sizeof(float);
	float * G, *G_last = NULL;
	hipMalloc(&G, gBytes);
	CUDA_CHECK;
	hipMemset(G, 0, gBytes);
	CUDA_CHECK;
	hipMalloc(&G_last, gBytes);
	CUDA_CHECK;
	hipMemset(G_last, 0, gBytes);
	CUDA_CHECK;

	float * err = NULL;
	hipMalloc(&err, sizeof(float));
	CUDA_CHECK;
	hipMemset(err, 0, sizeof(float));
	CUDA_CHECK;

	// copy data to device
	hipMemcpy(IL, imgInL, imgBytes, hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(IR, imgInR, imgBytes, hipMemcpyHostToDevice);
	CUDA_CHECK;

	// create kernel dimensions
	dim3 block2D(128, 1);
	dim3 grid2D((w + block2D.x - 1) / block2D.x,
			(h + block2D.y - 1) / block2D.y);

	dim3 block3D(64, 2, 1);
	dim3 grid3D((w + block3D.x - 1) / block3D.x,
			(h + block3D.y - 1) / block3D.y, (gc + block3D.z - 1) / block3D.z);

	dim3 block1D(256);
	dim3 grid1DP((w * h * gc * 3 + block1D.x - 1) / block1D.x);
	dim3 grid1DPhi((w * h * gc + block1D.x - 1) / block1D.x);

	// Actual Algorithm
	// Initialization works with 0s for P, only for Phi the first layer needs to be 1s
	// Thus run the projection once
	g_project_phi_d<<<grid3D, block3D>>>(Phi, w, h, gc);
	CUDA_CHECK;

	// Iterate until stopping criterion is reached
	int iterations = 0;
	while (1)
	{
		// Reset gradient and divergence
		hipMemset(Grad3_Phi, 0, pBytes);
		CUDA_CHECK;
		hipMemset(Div3_P, 0, phiBytes);
		CUDA_CHECK;

		// Calculate the divergence of P for the update step of phi
		g_div3<<<grid2D, block2D>>>(P, Div3_P, w, h, gc);
		CUDA_CHECK;

		// Update the Phi
		g_update_phi<<<grid2D, block2D>>>(Phi, Div3_P, w, h, gc, c.tau_p);
		CUDA_CHECK;

		// Make sure Phi is in the solution space (D)
		g_project_phi_d<<<grid2D, block2D>>>(Phi, w, h, gc);
		CUDA_CHECK;

		// Calculate the gradient in x, y, and gamma direction
		g_grad3<<<grid2D, block2D>>>(Phi, Grad3_Phi, w, h, gc);
		CUDA_CHECK;

		// Update the P
		g_update_p<<<grid2D, block2D>>>(P, Grad3_Phi, w, h, gc, c.tau_d);
		CUDA_CHECK;

		// Make sure P is in solution space (C)
		g_project_p_c<<<grid2D, block2D>>>(P, IL, IR, w, h, nc, gc, c.lambda,
				c.gamma_min);
		CUDA_CHECK;

		if (iterations > c.max_iterations)
			break;

		// TODO: convergence check via energy that is minimized, not via change of g
		// check convergence
		if (iterations % 1000 == 0)
		{
			// Save G of last convergence check
			hipMemcpy(G_last, G, gBytes, hipMemcpyDeviceToDevice);
			CUDA_CHECK;

			// Calculate the new G
			g_compute_g<<<grid2D, block2D>>>(Phi, G, w, h, c.gamma_min,
					c.gamma_max);
			CUDA_CHECK;

			hipMemset(err, 0, sizeof(float));
			CUDA_CHECK;

			g_squared_err_g<<<grid2D, block2D>>>(G, G_last, w, h, err);
			CUDA_CHECK;

			float err_host = 0.f;
			hipMemcpy(&err_host, err, sizeof(float), hipMemcpyDeviceToHost);
			CUDA_CHECK;

			cout << iterations << ": Error is " << err_host << endl;

			if (sqrt(err_host) < 0.01 || iterations > c.max_iterations)
				break;
		}

		iterations++;
	}

	// Move disparities from device to host
	hipMemcpy(imgOut, G, gBytes, hipMemcpyDeviceToHost);
	CUDA_CHECK;

	// show output image: first convert to interleaved opencv format from the layered raw array
	convert_layered_to_mat(mOut, imgOut);

	// free allocated arrays
	delete[] imgInL;
	delete[] imgInR;
	delete[] imgOut;

	hipFree(IL);
	CUDA_CHECK;
	hipFree(IR);
	CUDA_CHECK;
	hipFree(P);
	CUDA_CHECK;
	hipFree(Phi);
	CUDA_CHECK;
	hipFree(Grad3_Phi);
	CUDA_CHECK;
	hipFree(Div3_P);
	CUDA_CHECK;
	hipFree(G);
	CUDA_CHECK;
	hipFree(G_last);
	CUDA_CHECK;
	hipFree(err);
	CUDA_CHECK;

	return mOut;
}


cv::Mat adaptive_diffusion(const cv::Mat mDisparities, const cv::Mat mIn,
		const config c)
{
	cv::Mat mDiffused;

	// Width, height and channels of image
	int w, h, nc;
	get_dimensions(mIn, w, h, nc);

	// Convert to layered representation
	float *imgIn = new float[(size_t) w * h * nc];
	convert_mat_to_layered(imgIn, mIn);

	float *imgDisparities = new float[(size_t) w * h];
	convert_mat_to_layered(imgDisparities, mDisparities);

	// Input, Disparities, Output image
	float *In, *Disparities, *Depths, *Out = NULL;

	// Gradient in x,y direction, divergence
	float *Grad_x, *Grad_y, *Divergence = NULL;

	size_t nbytes = (size_t) (w * h * nc) * sizeof(float);
	size_t ndisparities = (size_t) (w * h) * sizeof(int);
	size_t ndepths = (size_t) (w * h) * sizeof(float);

	// Reserve space on device
	hipMalloc(&In, nbytes);
	CUDA_CHECK;
	hipMalloc(&Out, nbytes);
	CUDA_CHECK;
	hipMalloc(&Grad_x, nbytes);
	CUDA_CHECK;
	hipMalloc(&Grad_y, nbytes);
	CUDA_CHECK;
	hipMalloc(&Divergence, nbytes);
	CUDA_CHECK;
	hipMalloc(&Disparities, ndisparities);
	CUDA_CHECK;
	hipMalloc(&Depths, ndepths);
	CUDA_CHECK;

	hipMemset(In, 0, nbytes);
	CUDA_CHECK;
	hipMemset(Out, 0, nbytes);
	CUDA_CHECK;

	// Copy disparities to device
	hipMemcpy(Disparities, imgDisparities, ndisparities,
			hipMemcpyHostToDevice);
	CUDA_CHECK;

	// Copy image to device
	hipMemcpy(In, imgIn, nbytes, hipMemcpyHostToDevice);
	CUDA_CHECK;

	dim3 block2D(128, 1);
	dim3 grid2D((w + block2D.x - 1) / block2D.x,
			(h + block2D.y - 1) / block2D.y);

	dim3 block3D(64, 2, 1);
	dim3 grid3D((w + block3D.x - 1) / block3D.x,
			(h + block3D.y - 1) / block3D.y, (nc + block3D.z - 1) / block3D.z);

	// Compute the depth from the disparity values
	g_compute_depth<<<grid2D, block2D>>>(Disparities, Depths, w, h, c.baseline, c.focal_length, c.doffs);

	// ---- Calculate the G matrix
	float a = alpha(c.focal_length, 1.f, c.focal_plane);

	// Setup disparities as texture memory
	texRef.addressMode[0] = hipAddressModeClamp;// clamp x to border
	texRef.addressMode[1] = hipAddressModeClamp; // clamp y to border
	texRef.filterMode = hipFilterModeLinear; // linear interpolation
	texRef.normalized = false; // access as (x+0.5f,y+0.5f), not as ((x+0.5f)/w,(y+0.5f)/h)
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, &texRef, Disparities, &desc, w, h, sizeof(float));

	for (int i = 0; i < c.max_iterations; i++)
	{
		// reset the gradient/divergence data
		hipMemset(Grad_x, 0, nbytes);
		CUDA_CHECK;
		hipMemset(Grad_y, 0, nbytes);
		CUDA_CHECK;
		hipMemset(Divergence, 0, nbytes);
		CUDA_CHECK;

		//-- Gradient
		g_gradient<<<grid3D, block3D>>>(In, Grad_x, Grad_y, w, h, nc);
		CUDA_CHECK;

		// apply G matrix from texture memory
		g_apply_g<<<grid2D, block2D>>>(Grad_x, Grad_y, w, h, nc);
		CUDA_CHECK;

		// calculate divergence
		g_divergence<<<grid3D, block3D>>>(Grad_x, Grad_y, Divergence, w, h, nc);
		CUDA_CHECK;

		// do the update step
		g_update_step<<<grid3D, block3D>>>(In, Divergence, w, h, nc, c.tau);
		CUDA_CHECK;
	}

	// Copy result back to host
	hipMemcpy(imgIn, In, nbytes, hipMemcpyDeviceToHost);
	CUDA_CHECK;

	convert_layered_to_mat(mDiffused, imgIn);

	delete[] imgIn;
	delete[] imgDisparities;

	return mDiffused;
}

int main(int argc, char **argv)
{
	init_device();

	// Create empty config
	config c;

	read_parameters(c, argc, argv);

	// Load input images
	std::string imageL = c.image + "0.png", imageR = c.image + "1.png";

	cv::Mat mInL = load_image(imageL, c.gray);
	cv::Mat mInR = load_image(imageR, c.gray);

	// Get disparities from dataset or calculate
	cv::Mat mDisparities;
	if (c.disparities_from_file)
	{
		mDisparities = load_pfm(c.disparities);
	}
	else
	{
		mDisparities = calculate_disparities(c);
	}

	// Do anisotropic diffusion with the depth values
	cv::Mat mOut = adaptive_diffusion(mDisparities, mInL, c);

	showImage("Input", mInL, 100, 100);
	// normalize(mDisparities, mDisparities, 0.f, 1.f, cv::NORM_MINMAX, CV_32FC1);
	showImage("Disparities", mDisparities, 500, 100);

	showImage("Output", mOut, 100, 500);

	// wait for key inputs
	cv::waitKey(0);

	// close all opencv windows
	cvDestroyAllWindows();
	return 0;
}
