#include "hip/hip_runtime.h"
#include "util.h"
#include "usage.h"
#include "image.h"
#include "gradient.h"
#include "reduce.h"
#include "kernels.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include "assert.h"
using namespace std;

void check_Phi(float * Phi, int w, int h, int gc)
{
	hipDeviceSynchronize();

	float * phi_check = new float[w * h];
	for (int g = 0; g < gc; g++)
	{
		hipMemcpy(phi_check, &Phi[g * w * h], w * h * sizeof(float),
				hipMemcpyDeviceToHost);
		CUDA_CHECK;
		for (int i = 0; i < w * h; i++)
		{
			if (g == 0)
			{
				assert(phi_check[i] == 1.f);
			}
			else if (g == gc)
			{
				assert(phi_check[i] == 0.f);
			}
			else
			{
				assert(phi_check[i] <= 1.f && phi_check[i] >= 0.f);
			}
		}
	}
}

void check_P(float * P, float *Rho, int w, int h, int gc)
{
	hipDeviceSynchronize();

	float * p1_check = new float[w * h];
	float * p2_check = new float[w * h];
	float * p3_check = new float[w * h];
	float * rho = new float[w * h];

	for (int g = 0; g < gc; g++)
	{
		size_t ip1 = g * w * h;
		size_t ip2 = (1 * gc + g) * w * h;
		size_t ip3 = (2 * gc + g) * w * h;
		hipMemcpy(p1_check, &P[ip1], w * h * sizeof(float),
				hipMemcpyDeviceToHost);
		CUDA_CHECK;

		hipMemcpy(p2_check, &P[ip2], w * h * sizeof(float),
				hipMemcpyDeviceToHost);
		CUDA_CHECK;

		hipMemcpy(p3_check, &P[ip3], w * h * sizeof(float),
				hipMemcpyDeviceToHost);
		CUDA_CHECK;

		hipMemcpy(rho, &Rho[g * w * h], w * h * sizeof(float),
				hipMemcpyDeviceToHost);
		CUDA_CHECK;

		for (int i = 0; i < w * h; i++)
		{
			if (sqrtf(square(p1_check[i]) + square(p2_check[i])) > 1)
			{
				cout << "i: " << i << ", g: " << g << ": p1=" << p1_check[i]
						<< ", p2=" << p2_check[i] << endl;
			}
			if (fabs(p3_check[i]) > rho[i])
			{
				cout << "i: " << i << ", g: " << g << ": rho=" << rho[i]
						<< ", p3=" << p3_check[i] << endl;
			}
		}
	}
}

cv::Mat calculate_disparities(const config c, cv::Mat mDisparities)
{
	// define the range of gamma
	int gc = c.gamma_max - c.gamma_min;

	// image + 0 is left
	string imageL = c.image + "0.png";
	// image + 1 is right
	string imageR = c.image + "1.png";

	cv::Mat mInL = load_image(imageL, c.gray, c.max_w, c.max_h);
	cv::Mat mInR = load_image(imageR, c.gray, c.max_w, c.max_h);

	// Width, height and channels of image
	int w, h, nc;
	get_dimensions(mInL, mInR, w, h, nc);

	// Define output array, grayscale image of depth values
	cv::Mat mOut(h, w, CV_32FC1);

	// allocate raw input image array
	float *imgInL = new float[(size_t) w * h * nc];
	float *imgInR = new float[(size_t) w * h * nc];

	float *imgDisparities = new float[(size_t) w * h];

	// allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
	float *imgOut = new float[(size_t) w * h * mOut.channels()];

	// Init raw input image array
	convert_mat_to_layered(imgInL, mInL);
	convert_mat_to_layered(imgInR, mInR);

	convert_mat_to_layered(imgDisparities, mDisparities);

	// Allocate memory on device for images
	size_t imgBytes = w * h * nc * sizeof(float);
	float *IL, *IR = NULL;
	hipMalloc(&IL, imgBytes);
	CUDA_CHECK;
	hipMalloc(&IR, imgBytes);
	CUDA_CHECK;
	hipMemset(IL, 0, imgBytes);
	CUDA_CHECK;
	hipMemset(IR, 0, imgBytes);
	CUDA_CHECK;

	// for P (3 channels for p1..3) and Phi vectors
	size_t pBytes = w * h * gc * 3 * sizeof(float);
	size_t phiBytes = w * h * gc * sizeof(float);
	float *P, *Phi = NULL;
	hipMalloc(&P, pBytes);
	CUDA_CHECK;
	hipMalloc(&Phi, phiBytes);
	CUDA_CHECK;
	hipMemset(P, 0, pBytes);
	CUDA_CHECK;
	hipMemset(Phi, 0, phiBytes);
	CUDA_CHECK;

	// for grad3 of phi and div3 of p
	float * Grad3_Phi, *Div3_P = NULL;
	hipMalloc(&Grad3_Phi, pBytes);
	CUDA_CHECK;
	hipMalloc(&Div3_P, phiBytes);
	CUDA_CHECK;
	hipMemset(Grad3_Phi, 0, pBytes);
	CUDA_CHECK;
	hipMemset(Div3_P, 0, phiBytes);
	CUDA_CHECK;

	// for the error values Rho
	float * Rho = NULL;
	size_t rhoBytes = w * h * gc * sizeof(float);
	hipMalloc(&Rho, rhoBytes);
	CUDA_CHECK;
	hipMemset(Rho, 0, rhoBytes);
	CUDA_CHECK;

	// for the final depth values
	float * U = NULL;
	size_t uBytes = w * h * sizeof(float);
	hipMalloc(&U, uBytes);
	CUDA_CHECK;
	hipMemset(U, 0, uBytes);
	CUDA_CHECK;

	float * energy = NULL;
	hipMalloc(&energy, sizeof(float));
	CUDA_CHECK;
	hipMemset(energy, 0, sizeof(float));
	CUDA_CHECK;

	// copy data to device
	hipMemcpy(IL, imgInL, imgBytes, hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(IR, imgInR, imgBytes, hipMemcpyHostToDevice);
	CUDA_CHECK;

	// create kernel dimensions
	dim3 block2D(128, 1);
	dim3 grid2D((w + block2D.x - 1) / block2D.x,
			(h + block2D.y - 1) / block2D.y);

	dim3 block3D(64, 2, 1);
	dim3 grid3D((w + block3D.x - 1) / block3D.x,
			(h + block3D.y - 1) / block3D.y, (gc + block3D.z - 1) / block3D.z);

	dim3 block1D(256);
	dim3 grid1DP((w * h * gc * 3 + block1D.x - 1) / block1D.x);
	dim3 grid1DPhi((w * h * gc + block1D.x - 1) / block1D.x);

	// Actual Algorithm
	// Initialize Phi
	hipMemcpy(U, imgDisparities, uBytes, hipMemcpyHostToDevice);
	CUDA_CHECK;

	g_init_phi<<<grid2D, block2D>>>(Phi, U, w, h, gc);
	CUDA_CHECK;

	hipMemset(U, 0, uBytes);
	CUDA_CHECK;

	check_Phi(Phi, w, h, gc);

	for (int g = 0; g < gc; g++)
	{
		stringstream path;
		path << "phi/phi_00000_" << setfill('0') << setw(3) << g + c.gamma_min;
		save_from_GPU(path.str(), &Phi[g * w * h], w, h);
	}

	// Compute a global rho (that doesn't change...)
	g_compute_rho<<<grid2D, block2D>>>(IL, IR, Rho, w, h, nc, c.gamma_min,
			c.gamma_max, c.lambda);
	CUDA_CHECK;

	for (int g = 0; g < gc; g++)
	{
		stringstream path;
		path << "rho/rho_" << setfill('0') << setw(3) << g + c.gamma_min;
		save_from_GPU(path.str(), &Rho[g * w * h], w, h);
	}

	// Iterate until stopping criterion is reached
	int iterations = 1;
	while (1)
	{
		// Reset gradient and divergence
		hipMemset(Grad3_Phi, 0, pBytes);
		CUDA_CHECK;
		hipMemset(Div3_P, 0, phiBytes);
		CUDA_CHECK;

		// Calculate the divergence of P for the update step of phi
		g_div3<<<grid2D, block2D>>>(P, Div3_P, w, h, gc, c.dx, c.dy, c.dg);

		// Update the Phi
		g_update_phi<<<grid2D, block2D>>>(Phi, Div3_P, w, h, gc, c.tau_p);
		CUDA_CHECK;

		check_Phi(Phi, w, h, gc);

		// Calculate the gradient in x, y, and gamma direction
		g_grad3<<<grid2D, block2D>>>(Phi, Grad3_Phi, w, h, gc, c.dx, c.dy,
				c.dg);
		CUDA_CHECK;

		// Update the P
		g_update_p<<<grid2D, block2D>>>(P, Grad3_Phi, Rho, w, h, gc, c.tau_d);
		CUDA_CHECK;

		// check convergence
		if (iterations % (c.max_iterations / 2) == 0)
		{
			cout << "Iteration " << iterations << endl;
			/*
			 for (int g = 0; g < gc; g++)
			 {
			 stringstream path1, path2, path3;
			 path1 << "p/p1_" << setfill('0') << setw(5) <<  iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 path2 << "p/p2_" << setfill('0') << setw(5) <<  iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 path3 << "p/p3_" << setfill('0') << setw(5) << 	 iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 size_t ip1 = g * w * h;
			 size_t ip2 = (1 * gc + g) * w * h;
			 size_t ip3 = (2 * gc + g) * w * h;
			 save_from_GPU(path1.str(), &P[ip1], w, h);
			 save_from_GPU(path2.str(), &P[ip2], w, h);
			 save_from_GPU(path3.str(), &P[ip3], w, h);
			 }

			 for (int g = 0; g < gc; g++)
			 {
			 stringstream pathx, pathy, pathg;
			 pathx << "grad/gradx_" << setfill('0') << setw(5) <<  iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 pathy << "grad/grady_" << setfill('0') << setw(5) <<  iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 pathg << "grad/gradg_" << setfill('0') << setw(5) <<  iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 size_t ip1 = g * w * h;
			 size_t ip2 = (1 * gc + g) * w * h;
			 size_t ip3 = (2 * gc + g) * w * h;
			 save_from_GPU(pathx.str(), &Grad3_Phi[ip1], w, h);
			 save_from_GPU(pathy.str(), &Grad3_Phi[ip2], w, h);
			 save_from_GPU(pathg.str(), &Grad3_Phi[ip3], w, h);
			 }
			 */
			 for (int g = 0; g < gc; g++)
			 {
			 stringstream path;
			 path << "phi/phi_" << setfill('0') << setw(5) << iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 save_from_GPU(path.str(), &Phi[g * w * h], w, h);
			 }
			 /*
			 for (int g = 0; g < gc; g++)
			 {
			 stringstream path;
			 path << "div/div_" << setfill('0') << setw(5) << iterations << "_" << setfill('0') << setw(3) << g + c.gamma_min;
			 save_from_GPU(path.str(), &Div3_P[g * w * h], w, h);
			 }
			 */

			// Calculate the new G
			g_compute_u<<<grid2D, block2D>>>(Phi, U, w, h, c.gamma_min,
					c.gamma_max);
			CUDA_CHECK;

			hipMemset(energy, 0, sizeof(float));
			CUDA_CHECK;

			g_compute_energy<<<grid2D, block2D>>>(U, IL, IR, energy, w, h, nc,
					c.lambda);
			CUDA_CHECK;

			float energy_host = 0.f;
			hipMemcpy(&energy_host, energy, sizeof(float),
					hipMemcpyDeviceToHost);
			CUDA_CHECK;

			cout << iterations << ": Energy is " << energy_host << endl;

			if (energy_host < 0.01 || iterations >= c.max_iterations)
				break;
		}

		iterations++;
	}

	// Move disparities from device to host
	hipMemcpy(imgOut, U, uBytes, hipMemcpyDeviceToHost);
	CUDA_CHECK;

	// show output image: first convert to interleaved opencv format from the layered raw array
	convert_layered_to_mat(mOut, imgOut);

	// free allocated arrays
	delete[] imgInL;
	delete[] imgInR;
	delete[] imgOut;

	hipFree(IL);
	CUDA_CHECK;
	hipFree(IR);
	CUDA_CHECK;
	hipFree(P);
	CUDA_CHECK;
	hipFree(Phi);
	CUDA_CHECK;
	hipFree(Grad3_Phi);
	CUDA_CHECK;
	hipFree(Div3_P);
	CUDA_CHECK;
	hipFree(U);
	CUDA_CHECK;

	return mOut;
}

cv::Mat adaptive_diffusion(const cv::Mat mDisparities, const cv::Mat mIn,
		const config c)
{
	cout << "Depth Adaptive Diffusion" << endl;

	// Width, height and channels of image
	int w, h, nc;
	get_dimensions(mIn, w, h, nc);

	cv::Mat mDiffused = cv::Mat(h, w, CV_32FC3);

	// Convert to layered representation
	float *imgIn = new float[(size_t) w * h * nc];
	convert_mat_to_layered(imgIn, mIn);

	float *imgDisparities = new float[(size_t) w * h];
	convert_mat_to_layered(imgDisparities, mDisparities);

	// Input, Disparities, Output image
	float *In, *Out = NULL;
	size_t nbytes = (size_t) (w * h * nc) * sizeof(float);

	float *Disparities, *Depths = NULL;
	size_t ndisparities = (size_t) (w * h) * sizeof(float);
	size_t ndepths = (size_t) (w * h) * sizeof(float);

	// G matrix
	float *G = NULL;
	size_t ngbytes = (size_t) (w * h) * sizeof(float);

	// Gradient in x,y direction, divergence
	float *Grad_x, *Grad_y, *Divergence = NULL;

	// Reserve space on device
	hipMalloc(&In, nbytes);
	CUDA_CHECK;
	hipMalloc(&Out, nbytes);
	CUDA_CHECK;
	hipMalloc(&Grad_x, nbytes);
	CUDA_CHECK;
	hipMalloc(&Grad_y, nbytes);
	CUDA_CHECK;
	hipMalloc(&Divergence, nbytes);
	CUDA_CHECK;
	hipMalloc(&Disparities, ndisparities);
	CUDA_CHECK;
	hipMalloc(&Depths, ndepths);
	CUDA_CHECK;
	hipMalloc(&G, ngbytes);
	CUDA_CHECK;

	hipMemset(In, 0, nbytes);
	CUDA_CHECK;
	hipMemset(Out, 0, nbytes);
	CUDA_CHECK;
	hipMemset(G, 0, ngbytes);
	CUDA_CHECK;

	// Copy disparities to device
	hipMemcpy(Disparities, imgDisparities, ndisparities,
			hipMemcpyHostToDevice);
	CUDA_CHECK;

	// Copy image to device
	hipMemcpy(In, imgIn, nbytes, hipMemcpyHostToDevice);
	CUDA_CHECK;

	dim3 block2D(128, 1);
	dim3 grid2D((w + block2D.x - 1) / block2D.x,
			(h + block2D.y - 1) / block2D.y);

	dim3 block3D(64, 2, nc);
	dim3 grid3D((w + block3D.x - 1) / block3D.x,
			(h + block3D.y - 1) / block3D.y, (nc + block3D.z - 1) / block3D.z);

	// Compute the depth from the disparity values
	g_compute_depth<<<grid2D, block2D>>>(Disparities, Depths, w, h, c.baseline,
			c.focal_length, c.doffs);
	CUDA_CHECK;

	// Normalize to [0, 1]
	normalize(Depths, w, h, 0.f, 1.f);

	// ---- Calculate the G matrix
	g_compute_g_matrix<<<grid2D, block2D>>>(Depths, G, w, h, c.focal_plane,
			c.radius);
	CUDA_CHECK;

	// Normalize to [0, 1]
	normalize(G, w, h, 0.f, 1.f);

	save_from_GPU("depths", Depths, w, h);
	save_from_GPU("g", G, w, h);

	for (int i = 0; i < 15; i++)
	{
		// reset the gradient/divergence data
		hipMemset(Grad_x, 0, nbytes);
		CUDA_CHECK;
		hipMemset(Grad_y, 0, nbytes);
		CUDA_CHECK;
		hipMemset(Divergence, 0, nbytes);
		CUDA_CHECK;

		//-- Gradient
		g_gradient<<<grid3D, block3D>>>(In, Grad_x, Grad_y, w, h, nc);
		CUDA_CHECK;

		// apply G matrix from texture memory
		g_apply_g<<<grid2D, block2D>>>(Grad_x, Grad_y, G, w, h, nc);
		CUDA_CHECK;

		// calculate divergence
		g_divergence<<<grid3D, block3D>>>(Grad_x, Grad_y, Divergence, w, h, nc);
		CUDA_CHECK;

		// do the update step
		g_update_step<<<grid3D, block3D>>>(In, Divergence, w, h, nc, c.tau);
		CUDA_CHECK;
	}

	// Copy result back to host
	hipMemcpy(imgIn, In, nbytes, hipMemcpyDeviceToHost);
	CUDA_CHECK;

	convert_layered_to_mat(mDiffused, imgIn);

	// free memory
	hipFree(In);
	CUDA_CHECK;
	hipFree(G);
	CUDA_CHECK;
	hipFree(Grad_x);
	CUDA_CHECK;
	hipFree(Grad_y);
	CUDA_CHECK;
	hipFree(Divergence);
	CUDA_CHECK;
	hipFree(Disparities);
	CUDA_CHECK;
	hipFree(Depths);
	CUDA_CHECK;

	delete[] imgIn;
	delete[] imgDisparities;

	return mDiffused;
}

int main(int argc, char **argv)
{
	init_device();

	// Create empty config
	config c;

	read_parameters(c, argc, argv);

	// Load input images
	std::string imageL = c.image + "0.png", imageR = c.image + "1.png";

	cv::Mat mInL = load_image(imageL, c.gray, c.max_w, c.max_h);
	cv::Mat mInR = load_image(imageR, c.gray, c.max_w, c.max_h);

	int w, h, nc;
	get_dimensions(mInL, w, h, nc);

	cout << "Image Dimensions: " << w << "x" << h << "x" << nc << endl;

	// Get disparities from dataset or calculate
	cv::Mat mDisparities;
	if (c.disparities_from_file)
	{
		mDisparities = load_pfm(c.disparities, c.max_w, c.max_h);
	}
	else
	{
		cerr
				<< "ERROR: Call without ground truth not supported in this version!"
				<< endl;
		exit(1);
	}
	mDisparities = calculate_disparities(c, mDisparities);

	// Do anisotropic diffusion with the depth values
	cv::Mat mOut = adaptive_diffusion(mDisparities, mInL, c);

	showImage("Input", mInL, 100, 100);

	// Reduce range from [0, 255] to [0, 1]
	mDisparities /= 255.f;
	//normalize(mDisparities, mDisparities, 0.f, 1.f, cv::NORM_MINMAX, CV_32FC1);
	showImage("Disparities", mDisparities, 600, 100);
	save_image("disparities", mDisparities);

	showImage("Output", mOut, 100, 600);
	save_image("out", mOut);
	// wait for key inputs
	cv::waitKey(0);

	// close all opencv windows
	cvDestroyAllWindows();
	return 0;
}
