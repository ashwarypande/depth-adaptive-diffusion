#include "hip/hip_runtime.h"
#include "image.h"
#include <string>
#include <iostream>
#include <fstream>

using namespace std;
using namespace cv;

#define MAX_W 1024
#define MAX_H 768

// opencv helpers
void convert_layered_to_interleaved(float *aOut, const float *aIn, int w, int h,
		int nc)
{
	if (nc == 1)
	{
		memcpy(aOut, aIn, w * h * sizeof(float));
		return;
	}
	size_t nOmega = (size_t) w * h;
	for (int y = 0; y < h; y++)
	{
		for (int x = 0; x < w; x++)
		{
			for (int c = 0; c < nc; c++)
			{
				aOut[(nc - 1 - c) + nc * (x + (size_t) w * y)] = aIn[x
						+ (size_t) w * y + nOmega * c];
			}
		}
	}
}

void convert_layered_to_mat(Mat &mOut, const float *aIn)
{
	convert_layered_to_interleaved((float*) mOut.data, aIn, mOut.cols,
			mOut.rows, mOut.channels());
}

void convert_interleaved_to_layered(float *aOut, const float *aIn, int w, int h,
		int nc)
{
	if (nc == 1)
	{
		memcpy(aOut, aIn, w * h * sizeof(float));
		return;
	}
	size_t nOmega = (size_t) w * h;
	for (int y = 0; y < h; y++)
	{
		for (int x = 0; x < w; x++)
		{
			for (int c = 0; c < nc; c++)
			{
				aOut[x + (size_t) w * y + nOmega * c] = aIn[(nc - 1 - c)
						+ nc * (x + (size_t) w * y)];
			}
		}
	}
}

void convert_mat_to_layered(float *aOut, const Mat &mIn)
{
	convert_interleaved_to_layered(aOut, (float*) mIn.data, mIn.cols, mIn.rows,
			mIn.channels());
}

void showImage(string title, const Mat &mat, int x, int y)
{
	const char *wTitle = title.c_str();
	namedWindow(wTitle, CV_WINDOW_AUTOSIZE);
	cvMoveWindow(wTitle, x, y);
	imshow(wTitle, mat);
}

void showHistogram256(const char *windowTitle, int *histogram, int windowX,
		int windowY)
{
	const int nbins = 256;
	Mat canvas = Mat::ones(125, 512, CV_8UC3);

	float hmax = 0;
	for (int i = 0; i < nbins; ++i)
		hmax = max((int) hmax, histogram[i]);

	for (int j = 0, rows = canvas.rows; j < nbins - 1; j++)
	{
		for (int i = 0; i < 2; ++i)
			line(canvas, Point(j * 2 + i, rows),
					Point(j * 2 + i, rows - (histogram[j] * 125.0f) / hmax),
					Scalar(255, 128, 0), 1, 8, 0);
	}

	showImage(windowTitle, canvas, windowX, windowY);
}

// adding Gaussian noise
float noise(float sigma)
{
	float x1 = (float) rand() / RAND_MAX;
	float x2 = (float) rand() / RAND_MAX;
	return sigma * sqrtf(-2 * log(std::max(x1, 0.000001f)))
			* cosf(2 * M_PI * x2);
}

void addNoise(Mat &m, float sigma)
{
	float *data = (float*) m.data;
	int w = m.cols;
	int h = m.rows;
	int nc = m.channels();
	size_t n = (size_t) w * h * nc;
	for (size_t i = 0; i < n; i++)
	{
		data[i] += noise(sigma);
	}
}

void downsample(Mat &mIn, int max_w, int max_h)
{
	float factor = 0.f;
	// downsample if bigger than MAX_W or MAX_H
	if (mIn.cols > max_w)
		factor = (float) max_w / mIn.cols;

	if (mIn.rows > MAX_H)
		factor = min((float) max_h / mIn.rows, factor);

	if (factor > 0)
		resize(mIn, mIn, Size(), factor, factor, INTER_AREA);
}


Mat load_image(const std::string image, bool gray)
{
	// Load the input image using opencv
	// (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
	Mat mIn = imread(image.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
	// check
	if (mIn.data == NULL)
	{
		cerr << "ERROR: Could not load image " << image << endl;
		exit(1);
	}

	mIn.convertTo(mIn, CV_32F);
	mIn /= 255.f;

	// cout << "Original image is " << mIn.cols << " x " << mIn.rows << " x " << mIn.channels() << endl;

	downsample(mIn, MAX_W, MAX_H);

	return mIn;
}


/**
 * Implementation reference:
 * https://github.com/antoinetlc/PFM_ReadWrite/blob/master/PFMReadWrite.cpp
 */
Mat load_pfm(const std::string image)
{
	// Open image as binary filestream
	ifstream file(image.c_str(), ios::in | ios::binary);

	Mat mDisparities;

	// check if opened correctly
	if (file)
	{
		// filetype plus 0x0a Unix return
		char type[3];
		file.read(type, 3 * sizeof(char));

		// width and height
		unsigned int width = 0, height = 0;
		file >> width >> height;

		// 0x0a Unix return
		char eol;
		file.read(&eol, sizeof(char));

		int channels = 0;
		// type[1] indicates number of channels
		if (type[1] == 'F')
		{
			mDisparities = Mat(height, width, CV_32FC3);
			channels = 3;
		}
		else if (type[1] == 'f')
		{
			mDisparities = Mat(height, width, CV_32FC1);
			channels = 1;
		}

		// endianess plus 0x0a Unix return
		char byteorder[4];
		file.read(byteorder, 4 * sizeof(char));

		// read until pixels start
		char returnchar = ' ';
		while (returnchar != 0x0a)
		{
			file.read(&returnchar, sizeof(char));
		}

		// read all pixel to matrix
		float *color = new float[channels];
		for (int y = 0; y < height; y++)
		{
			for (int x = 0; x < width; x++)
			{
				// next pixel, values from 0-255 (char)
				file.read((char *) color, channels * sizeof(float));
				if (channels == 3)
				{
					// OpenCV stores color as BGR, pfm is RGB
					mDisparities.at<Vec3f>((height - 1) - y, x) = Vec3f(
							color[2], color[1], color[0]);
				}
				else if (channels == 1)
				{
					mDisparities.at<float>((height - 1) - y, x) = color[0];
				}
			}
		}

		// tidy up
		delete[] color;
	}
	else
	{
		cerr << "Unable ot open file " << image << endl;
		exit(1);
	}

	// close filestream
	file.close();

	// Change range (0-255) to (0-1)
	mDisparities /= 255.f;

	// downsample to max resolution
	downsample(mDisparities, MAX_W, MAX_H);

	return mDisparities;
}

void get_dimensions(const Mat &m1, const Mat &m2, int &w, int &h, int &nc)
{
	w = m1.cols;        // width
	h = m1.rows;        // height
	nc = m1.channels(); // number of channels

	// check if dimensions match
	if (w != m2.cols || h != m2.rows || nc != m2.channels())
	{
		cerr << "ERROR: image dimensions mismatch!" << endl;
		exit(1);
	}

}

void get_dimensions(const Mat &m, int &w, int &h, int &nc)
{
	w = m.cols;        // width
	h = m.rows;        // height
	nc = m.channels(); // number of channels
}


void save_image(string image_name, Mat &mOut)
{
	// save input and result
	imwrite(image_name + ".png", mOut * 255.f);
}

